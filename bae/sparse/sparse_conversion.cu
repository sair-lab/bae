#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <ATen/cuda/detail/DeviceThreadHandles.h>

#include <pybind11/pybind11.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
    }                                                                          \
}

torch::Tensor gebsr2csr_impl(torch::Tensor A) {
    TORCH_CHECK(A.layout() == torch::kSparseBsr, "A must be a BSR matrix");
    int *bsrRowPtrA, *bsrColIndA;
    torch::Tensor crow_a = A.crow_indices();
    torch::Tensor col_a = A.col_indices();
    if (crow_a.dtype() != torch::kInt32) {
        crow_a = crow_a.to(torch::kInt32);
        col_a = col_a.to(torch::kInt32);
    }
    // std::cout << "gebsr2csr" << std::endl << std::flush;
    bsrRowPtrA = crow_a.data<int>();
    bsrColIndA = col_a.data<int>();

    int m = A.size(-2);
    int n = A.size(-1);
    int rowBlockDim = A.values().size(-2);
    int colBlockDim = A.values().size(-1);
    int mb = m / rowBlockDim;
    int nb = n / colBlockDim;
    int nnzb = col_a.size(0); // number of blocks
    int nnz  = nnzb * rowBlockDim * colBlockDim; // number of elements

    torch::Tensor crow_c = torch::empty({m+1}, crow_a.options());
    int *csrRowPtrC = crow_c.data_ptr<int>();
    torch::Tensor col_c = torch::empty({nnz}, crow_a.options());
    int *csrColIndC = col_c.data_ptr<int>();
    torch::Tensor val_c = torch::empty({nnz}, A.values().options());
    hipsparseHandle_t handle = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipsparseMatDescr_t descrC = NULL;

    // Create matrix descriptors
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));
    auto stream = c10::cuda::getCurrentCUDAStream();
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUSPARSE(hipsparseSetStream(handle, stream));
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
    AT_DISPATCH_FLOATING_TYPES(A.values().scalar_type(), "creating_desc", ([&] {
        const scalar_t *bsrValA = A.values().data_ptr<scalar_t>();
        scalar_t *csrValC = val_c.data_ptr<scalar_t>();
        if (std::is_same<scalar_t, float>::value) {
            CHECK_CUSPARSE( hipsparseSgebsr2csr(handle, dir, mb, nb,
                descrA,
                (float *)bsrValA, bsrRowPtrA, bsrColIndA,
                rowBlockDim, colBlockDim,
                descrC,
                (float *)csrValC, csrRowPtrC, csrColIndC) )
        } else if (std::is_same<scalar_t, double>::value) {
            CHECK_CUSPARSE( hipsparseDgebsr2csr(handle, dir, mb, nb,
                descrA,
                (double *)bsrValA, bsrRowPtrA, bsrColIndA,
                rowBlockDim, colBlockDim,
                descrC,
                (double *)csrValC, csrRowPtrC, csrColIndC) )
        }
    }));

    CHECK_CUSPARSE( hipsparseDestroyMatDescr(descrA) );
    CHECK_CUSPARSE( hipsparseDestroyMatDescr(descrC) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );
    
    return at::_sparse_compressed_tensor_unsafe(crow_c, col_c, val_c, {m, n}, at::TensorOptions().dtype(val_c.dtype()).device(val_c.device()).layout(at::kSparseCsr));


}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("gebsr2csr", &gebsr2csr_impl, "Custom gebsr2csr function");
}
